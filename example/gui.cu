#include <filesystem/path.h>
#include <neural-graphics-primitives/testbed.h>
#include <tiny-cuda-nn/common.h>

#include <args/args.hxx>

#include <png.h>  // Add this at the top

void save_png(const std::string& filename, vec4 *image, size_t width, size_t height) {
    FILE* fp = fopen(filename.c_str(), "wb");
    if (!fp) {
        throw std::runtime_error("Failed to open PNG file for writing.");
    }

    png_structp png = png_create_write_struct(PNG_LIBPNG_VER_STRING, nullptr, nullptr, nullptr);
    png_infop info = png_create_info_struct(png);

    if (!png || !info) {
        fclose(fp);
        throw std::runtime_error("Failed to create PNG structs.");
    }

    if (setjmp(png_jmpbuf(png))) {
        png_destroy_write_struct(&png, &info);
        fclose(fp);
        throw std::runtime_error("libpng error during PNG creation.");
    }

    png_init_io(png, fp);
    png_set_IHDR(
        png,
        info,
        width,
        height,
        8,                         // Bit depth
        PNG_COLOR_TYPE_RGB,        // No alpha
        PNG_INTERLACE_NONE,
        PNG_COMPRESSION_TYPE_BASE,
        PNG_FILTER_TYPE_BASE
    );
    png_write_info(png, info);

    // Convert float vec4s to uint8 RGB
    std::vector<uint8_t> row_data(width * 3);
    for (size_t y = 0; y < height; y++) {
        for (size_t x = 0; x < width; x++) {
            size_t i = y * width + x;
            vec4 c = image[i];
            row_data[x * 3 + 0] = std::clamp(c.r * 255.0f, 0.0f, 255.0f);
            row_data[x * 3 + 1] = std::clamp(c.g * 255.0f, 0.0f, 255.0f);
            row_data[x * 3 + 2] = std::clamp(c.b * 255.0f, 0.0f, 255.0f);
        }
        png_write_row(png, row_data.data());
    }

    png_write_end(png, nullptr);
    png_destroy_write_struct(&png, &info);
    fclose(fp);
}

using namespace args;
using namespace ngp;
using namespace std;


int main()
{
  Testbed testbed;
  testbed.m_train = true;
  testbed.init_window(500, 500);
  testbed.frame();
  testbed.load_snapshot(static_cast<fs::path>(
      std::string("/home/quang/Quang/projects/fns/build/_deps/ingp-src/data/"
                  "nerf/fox_small/base.ingp")));

  testbed.frame();

  vec4* device_ptr =
      testbed.m_views.front().render_buffer.get()->frame_buffer();
  // vec4 * device_ptr = testbed.m_views.front().device->render_buffer_view().frame_buffer;
  size_t frame_width = testbed.m_views.front().full_resolution.x;
  size_t frame_height = testbed.m_views.front().full_resolution.y;

  vec4 *host_buffer = (vec4 *)malloc(sizeof(vec4) * frame_width * frame_height);

  hipMemcpy(host_buffer,
             device_ptr,
             sizeof(vec4) * frame_width * frame_height,
             hipMemcpyDeviceToHost);

  save_png("output.png", host_buffer, frame_width, frame_height);
  free(host_buffer);
  return 0;
}
